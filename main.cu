#include "hip/hip_runtime.h"
#include "stdGL.h"
#include <vector>
#include <iostream>
#include "objects.h"
#include "shader.h"
#include "hip/hip_vector_types.h"

#include <SDL2/SDL.h>
#include <SDL2/SDL_opengl.h>
//#include <SDL2/SDL_image.h>

using namespace std;

//GLOBAL VARIABLES//
//running or not
bool quit = false;
int Pause = 0;

//Window Size
int w = 1920;
int h = 1080;

//eye position and orientation
double ex = 0;
double ey = 0;
double ez = 0;
double zoom = 64;
double dzoom = 0;
double th = 0;
double ph = 0;
double dph = 0;
double dth = 0;

//Textures
unsigned int starTexture = 0;

//Shaders
int shader = 0;
int pixlight = 0;
//int textures = 0;
//int test = 0;

//Simulation Timestep
const float dt = 1.0/64.0;
//const float dt = 0.125;
bool changed = false;

//Cuda random state
hiprandState_t* curandstate;

// Array Sizes
//const int N = pow(2,20);
const int N = 1024*64;
const int M = 64;
int ping = 0;
int pong = 1;

float zeros[M*M*M*4] = {0.0};
float ones[M*M*M]    = {1.0};

//Particle Arrays
float* verts  = NULL;
float* pvels  = NULL;
float* times  = NULL;
float* colors = NULL;

float* dverts = NULL;
float* dpvels = NULL;
float* dtimes = NULL;
float* dcolors= NULL;

//Grid Arrays
float* h_gvels  = NULL;
//float* h_gtemp  = NULL;
//float* h_gdens  = NULL;

float4* d_gvels[2] = {NULL};
//float*  d_gtemp[2] = {NULL};
//float*  d_gdens[2] = {NULL};
//float*  d_gpres[2] = {NULL};
//float*  d_diverge  =  NULL;

//User Interaction
bool rotating = false;
bool flame_moving = false;
bool fieldlines = false;
bool stepmode = false;
bool gpu = true;
float flame_x = M/2.0;
float flame_y = M/2.0;
float tick_period = 4.0;
float decay_rate = 0.008;
//float flame_z = 0.0;

////////////////////
//functions that are called ahead of when they're defined
//because C
void reshape(int width, int height);
void keyboard(const Uint8* state);

///////// CUDA Functions //////////

//  Arrays
// Grid [MxMxM]
//density
//temperature
//velocity
// Particles [N]
//position
//velocity
//time
//color

//typedef hipTextureObject_t hipTextureObject_t;
//typedef surface<void,cudaSurfaceType3D> surface<void,cudaSurfaceType3D>;

// non-texture-memory texture lookup function
__device__ float4 tex3d(float4* tex, float i, float j, float k, int s_i, int s_j, int s_k) {
  //int r1 = floor(r); r1 = r1%s_r;
  //int r2 = ceil(r);  r2 = r2%s_r;
  //int s1 = floor(s); s1 = s1%s_s;
  //int s2 = ceil(s);  s2 = s2%s_s;
  //int t1 = floor(t); t1 = t1%s_t;
  //int t2 = ceil(t);  t2 = t2%s_t;
  i = clamp(i, 0.0, s_i-1.0);
  j = clamp(j, 0.0, s_j-1.0);
  k = clamp(k, 0.0, s_k-1.0);
  int i1 = floor(i);
  int i2 = ceil (i);
  int j1 = floor(j);
  int j2 = ceil (j);
  int k1 = floor(k);
  int k2 = ceil (k);

  //if (t1 == 0 || t2 == s_t-1)
  //  return 0.0;

  float4 a = tex[i1*s_j*s_k + j1*s_k + k1];
  float4 b = tex[i2*s_j*s_k + j1*s_k + k1];
  float4 c = tex[i1*s_j*s_k + j2*s_k + k1];
  float4 d = tex[i2*s_j*s_k + j2*s_k + k1];
  float4 e = tex[i1*s_j*s_k + j1*s_k + k2];
  float4 f = tex[i2*s_j*s_k + j1*s_k + k2];
  float4 g = tex[i1*s_j*s_k + j2*s_k + k2];
  float4 h = tex[i2*s_j*s_k + j2*s_k + k2];
  return trilerp(a,b,c,d,e,f,g,h, i-i1,j-j1,k-k1);
}
__device__ float tex3d(float* tex, float i, float j, float k, int s_i, int s_j, int s_k) {
  //int r1 = floor(r); r1 = r1%s_r;
  //int r2 = ceil(r);  r2 = r2%s_r;
  //int s1 = floor(s); s1 = s1%s_s;
  //int s2 = ceil(s);  s2 = s2%s_s;
  //int t1 = floor(t); t1 = t1%s_t;
  //int t2 = ceil(t);  t2 = t2%s_t;
  i = clamp(i, 0.0, s_i-1.0);
  j = clamp(j, 0.0, s_j-1.0);
  k = clamp(k, 0.0, s_k-1.0);
  int i1 = floor(i);
  int i2 = ceil (i);
  int j1 = floor(j);
  int j2 = ceil (j);
  int k1 = floor(k);
  int k2 = ceil (k);

  //if (t1 == 0 || t2 == s_t-1)
  //  return 0.0;

  float a = tex[i1*s_j*s_k + j1*s_k + k1];
  float b = tex[i2*s_j*s_k + j1*s_k + k1];
  float c = tex[i1*s_j*s_k + j2*s_k + k1];
  float d = tex[i2*s_j*s_k + j2*s_k + k1];
  float e = tex[i1*s_j*s_k + j1*s_k + k2];
  float f = tex[i2*s_j*s_k + j1*s_k + k2];
  float g = tex[i1*s_j*s_k + j2*s_k + k2];
  float h = tex[i2*s_j*s_k + j2*s_k + k2];
  return trilerp(a,b,c,d,e,f,g,h, i-i1,j-j1,k-k1);
}

__device__ void set_bnd(float4* vels) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z*blockDim.z + threadIdx.z;

  // Boundary conditions:
  // X: reflect
  // Y: reflect
  // Z: reflect
  if (i == 0) {
    float4 src = vels[(i+1)*M*M + j*M + k];
    vels[i*M*M + j*M + k] = make_float4(-src.x, src.y, src.z, src.w);
    //vels[i*M*M + j*M + k] = make_float4(0.0, 0.0, 0.0, src.w);
  }
  else if (i == M-1) {
    float4 src = vels[(i-1)*M*M + j*M + k];
    vels[i*M*M + j*M + k] = make_float4(-src.x, src.y, src.z, src.w);
    //vels[i*M*M + j*M + k] = make_float4(0.0, 0.0, 0.0, src.w);
  }
  if (j == 0) {
    float4 src = vels[i*M*M + (j+1)*M + k];
    vels[i*M*M + j*M + k] = make_float4(src.x, -src.y, src.z, src.w);
    //vels[i*M*M + j*M + k] = make_float4(0.0, 0.0, 0.0, src.w);
  }
  else if (j == M-1) {
    float4 src = vels[i*M*M + (j-1)*M + k];
    vels[i*M*M + j*M + k] = make_float4(src.x, -src.y, src.z, src.w);
    //vels[i*M*M + j*M + k] = make_float4(0.0, 0.0, 0.0, src.w);
  }
  if (k == 0) {
    float4 src = vels[i*M*M + j*M + (k+1)];
    vels[i*M*M + j*M + k] = make_float4(src.x, src.y, -src.z, src.w);
    //vels[i*M*M + j*M + k] = make_float4(0.0, 0.0, 0.0, src.w);
  }
  else if (k == M-1) {
    float4 src = vels[i*M*M + j*M + (k-1)];
    vels[i*M*M + j*M + k] = make_float4(src.x, src.y, -src.z, src.w);
    //vels[i*M*M + j*M + k] = make_float4(0.0, 0.0, 0.0, src.w);
  }
}

//__device__ void lin_solv(float4* x, float4* x0, float a, float c) {
//  int i = blockIdx.x*blockDim.x + threadIdx.x;
//  int j = blockIdx.y*blockDim.y + threadIdx.y;
//  int k = blockIdx.z*blockDim.z + threadIdx.z;
//
//// TODO: Enable cooperative syncing iff block-edges become noticable or incompressibility is broken
////  cooperative_groups::grid_group g = cooperative_groups::this_grid();
//
//  float cc = 1.0/c;
//  if (i > 0 && i < M-1 &&
//      j > 0 && j < M-1 &&
//      k > 0 && k < M-1) {
//    for (int iter=0; iter < 16; ++iter) {
//      x[i*M*M + j*M + k] =
//        (x0[i*M*M + j*M + k]
//        + a*(x[(i+1)*M*M + j*M + k] + x[(i-1)*M*M + j*M + k]
//           + x[i*M*M + (j+1)*M + k] + x[i*M*M + (j-1)*M + k]
//           + x[i*M*M + j*M + (k+1)] + x[i*M*M + j*M + (k-1)])
//        )*cc;
//      set_bnd(x);
////      g.sync();
//    }
//  }
//}

__global__ void diffuse(float4* x, float4* x0, float viscosity) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z*blockDim.z + threadIdx.z;

  float a = dt * viscosity;// * (M-2)*(M-2)*(M-2);
  float cc = 1.0/(1.0+6.0*a);
  if (i > 0 && i < M-1 &&
      j > 0 && j < M-1 &&
      k > 0 && k < M-1) {
    x[i*M*M + j*M + k] =
      (x0[i*M*M + j*M + k] +
          a*(x0[(i+1)*M*M + j*M + k] + x0[(i-1)*M*M + j*M + k]
           + x0[i*M*M + (j+1)*M + k] + x0[i*M*M + (j-1)*M + k]
           + x0[i*M*M + j*M + (k+1)] + x0[i*M*M + j*M + (k-1)])
      )*cc;
  }
  set_bnd(x);
}

__global__ void pressure(float4* vels, float4* vels0) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z*blockDim.z + threadIdx.z;

  if (i > 0 && i < M-1 &&
      j > 0 && j < M-1 &&
      k > 0 && k < M-1) {
    // collect neighboring densities
    float p_x0 = vels0[(i-1)*M*M + j*M + k].w;
    float p_x1 = vels0[(i+1)*M*M + j*M + k].w;
    float p_y0 = vels0[i*M*M + (j-1)*M + k].w;
    float p_y1 = vels0[i*M*M + (j+1)*M + k].w;
    float p_z0 = vels0[i*M*M + j*M + (k-1)].w;
    float p_z1 = vels0[i*M*M + j*M + (k+1)].w;
    // collect neighboring velocities
    float v_x0 = vels0[(i-1)*M*M + j*M + k].x;
    float v_x1 = vels0[(i+1)*M*M + j*M + k].x;
    float v_y0 = vels0[i*M*M + (j-1)*M + k].y;
    float v_y1 = vels0[i*M*M + (j+1)*M + k].y;
    float v_z0 = vels0[i*M*M + j*M + (k-1)].z;
    float v_z1 = vels0[i*M*M + j*M + (k+1)].z;
    // apply net pressure force
    float d_x = 5.0*(p_x0 - p_x1);
    float d_y = 5.0*(p_y0 - p_y1);
    float d_z = 5.0*(p_z0 - p_z1);
    // and add vertical buoyancy force
    //float p_b = vels0[i*M*M + j*M + k].w - 0.16666*(p_x0 + p_x1 + p_y0 + p_y1 + p_z0 + p_z1);
    //float p_b = 0.0;

    //float buoy = 1.0;
    //float a = dt;
    //float a = 5.0;

    // modify pressure based on net velocity
    float d_p = 1.0 * (v_x0 - v_x1
                     + v_y0 - v_y1
                     + v_z0 - v_z1);

    vels[i*M*M + j*M + k].x = vels0[i*M*M + j*M + k].x + dt*d_x;
    vels[i*M*M + j*M + k].y = vels0[i*M*M + j*M + k].y + dt*d_y;
    vels[i*M*M + j*M + k].z = vels0[i*M*M + j*M + k].z + dt*d_z;
    // small velocity decay to naturally trend back toward zero instead of infinity over time
    vels[i*M*M + j*M + k] *= 0.999;
    vels[i*M*M + j*M + k].w = d_p;
  }

  set_bnd(vels);
}

__global__ void project(float4* vels) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z*blockDim.z + threadIdx.z;

  // find divergences of adjacent cells
  float dx0 = 0.0;
  if (i > 1 && j > 0 && j < M-1 && k > 0 && k < M-1) {
    dx0 = 0.16666*( - vels[(i-2)*M*M + j*M + k].x
                    + vels[i*M*M + j*M + k].x
                    - vels[(i-1)*M*M + (j-1)*M + k].y
                    + vels[(i-1)*M*M + (j+1)*M + k].y
                    - vels[(i-1)*M*M + j*M + (k-1)].z
                    + vels[(i-1)*M*M + j*M + (k+1)].z);
  }
  float dx1 = 0.0;
  if (i < M-2 && j > 0 && j < M-1 && k > 0 && k < M-1) {
    dx1 = 0.16666*( - vels[i*M*M + j*M + k].x
                    + vels[(i+2)*M*M + j*M + k].x
                    - vels[(i+1)*M*M + (j-1)*M + k].y
                    + vels[(i+1)*M*M + (j+1)*M + k].y
                    - vels[(i+1)*M*M + j*M + (k-1)].z
                    + vels[(i+1)*M*M + j*M + (k+1)].z);
  }
  float dy0 = 0.0;
  if (i > 0 && i < M-1 && j > 1 && k > 0 && k < M-1) {
    dy0 = 0.16666*( - vels[(i-1)*M*M + (j-1)*M + k].x
                    + vels[(i+1)*M*M + (j-1)*M + k].x
                    - vels[i*M*M + (j-2)*M + k].y
                    + vels[i*M*M + j*M + k].y
                    - vels[i*M*M + (j-1)*M + (k-1)].z
                    + vels[i*M*M + (j-1)*M + (k+1)].z);
  }
  float dy1 = 0.0;
  if (i > 0 && i < M-1 && j < M-2 && k > 0 && k < M-1) {
    dy1 = 0.16666*( - vels[(i-1)*M*M + (j+1)*M + k].x
                    + vels[(i+1)*M*M + (j+1)*M + k].x
                    - vels[i*M*M + j*M + k].y
                    + vels[i*M*M + (j+2)*M + k].y
                    - vels[i*M*M + (j+1)*M + (k-1)].z
                    + vels[i*M*M + (j+1)*M + (k+1)].z);
  }
  float dz0 = 0.0;
  if (i > 0 && i < M-1 && j > 0 && j < M-1 && k > 1) {
    dz0 = 0.16666*( - vels[(i-1)*M*M + j*M + (k-1)].x
                    + vels[(i+1)*M*M + j*M + (k-1)].x - vels[i*M*M + (j-1)*M + (k-1)].y
                    + vels[i*M*M + (j+1)*M + (k-1)].y
                    - vels[i*M*M + j*M + (k-2)].z
                    + vels[i*M*M + j*M + k].z);
  }
  float dz1 = 0.0;
  if (i > 0 && i < M-1 && j > 0 && j < M-1 && k < M-2) {
    dz1 = 0.16666*( - vels[(i-1)*M*M + j*M + (k+1)].x
                    + vels[(i+1)*M*M + j*M + (k+1)].x
                    - vels[i*M*M + (j-1)*M + (k+1)].y
                    + vels[i*M*M + (j+1)*M + (k+1)].y
                    - vels[i*M*M + j*M + k].z
                    + vels[i*M*M + j*M + (k+2)].z);
  }

  // subtract pressure vectors from velocities
  vels[i*M*M + j*M + k].x -= 0.5*(dx1 - dx0);
  vels[i*M*M + j*M + k].y -= 0.5*(dy1 - dy0);
  vels[i*M*M + j*M + k].z -= 0.5*(dz1 - dz0);

  set_bnd(vels);
}

__global__ void balance(float4* vels) {//, float4* vels0) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z*blockDim.z + threadIdx.z;

  //float sum_pres = 100.0;
  float sum_pres = 1.0;

  //for (int I=1; I < M*M*M; ++I) {
  //  sum_pres += vels0[I].w;
  //}
  sum_pres = sum_pres / (M*M*M);
  vels[i*M*M + j*M + k].w -= sum_pres;

  set_bnd(vels);
}

__global__ void advect(float4* vels_out, float4* vels_in, float* verts, float* times) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j = blockIdx.y*blockDim.y + threadIdx.y;
  int k = blockIdx.z*blockDim.z + threadIdx.z;

  float fi = i - dt*vels_in[i*M*M + j*M + k].x;
  float fj = j - dt*vels_in[i*M*M + j*M + k].y;
  float fk = k - dt*vels_in[i*M*M + j*M + k].z;

  vels_out[i*M*M + j*M + k] = tex3d(vels_in, fi, fj, fk, M,M,M);

  //// Add impulse based on heat buoyancy from the particles
  //float impulse = 0.0;
  //for (int n=0; n<N; ++n) {
  //  float3 vert = make_float3(verts[n*3], verts[n*3+1], verts[n*3+2]);
  //  float dx = vert.x-i;
  //  float dy = vert.y-j;
  //  float dz = vert.z-k;
  //  if (abs(dx) < 1.0 && abs(dy) < 1.0 && abs(dz) < 1.0) {
  //    float dist = sqrt(dx*dx + dy*dy + dz*dz);
  //    impulse += (1.0-dist) * times[n];
  //  }
  //}
  //vels_out[i*M*M + j*M + k].z += 4.0*impulse/N;

  //if (abs(i - flame_x) <= 1.0 && abs(j - flame_y) <= 1.0 && k/2 == M/8) {
  //  vels_out[i*M*M + j*M + k].x = 0.0;
  //  vels_out[i*M*M + j*M + k].y = 0.0;
  //  vels_out[i*M*M + j*M + k].z = 0.5;
  //}

  set_bnd(vels_out);
}

__global__ void pingpong(float4* x, float4* x0) {
  // this has better performance than memcpy
  int I = blockIdx.x*blockDim.x + threadIdx.x;
  if (I < M*M*M) x[I] = x0[I];
}

__global__ void init_rand_state(hiprandState_t* curandstate) {
  int I = blockIdx.x*blockDim.x + threadIdx.x;
  hiprand_init(1729, I, 0, &curandstate[I]);
}

__global__ void pstep(float4* gvels, float* verts, float* times, float* colors, hiprandState_t* curandstate, float flame_x, float flame_y, float decay_rate) {
  // times index
  int I = blockIdx.x*blockDim.x + threadIdx.x;
  // verts & colors index
  int i = I * 3;
  // texture lookup of velocity at the particle's location
  float4 V = tex3d(gvels, verts[i], verts[i+1], verts[i+2], M,M,M);
  verts[i  ] += V.x;
  verts[i+1] += V.y;
  verts[i+2] += V.z;

  colors[i  ] = sqrt(times[I]);
  colors[i+1] = max(times[I]/1.125f, 0.0f);
  colors[i+2] = pow(times[I],2.0f)/2;
  //colors[i  ] = max(0.2, abs(V.x));
  //colors[i+1] = max(0.2, abs(V.y));
  //colors[i+2] = max(0.2, abs(V.z));
  times[I] -= decay_rate;
  if (times[I] < 0.0f) {
    times[I]  += 1.0f;
    hiprandState_t localstate0 = curandstate[I+0];
    hiprandState_t localstate1 = curandstate[I+1];
    hiprandState_t localstate2 = curandstate[I+2];
    verts[i  ] = (hiprand_normal(&localstate0)) + flame_x;
    verts[i+1] = (hiprand_normal(&localstate1)) + flame_y;
    verts[i+2] = (hiprand_normal(&localstate2)) + M/4;
    curandstate[I+0] = localstate0;
    curandstate[I+1] = localstate1;
    curandstate[I+2] = localstate2;
  }

  // Add impulse to gvels based on particle temperature
  //    float dist = sqrt(dx*dx + dy*dy + dz*dz);
  //    impulse += (1.0-dist) * times[n];
  //  }
  //}
  //vels_out[i*M*M + j*M + k].z += 4.0*impulse/N;
  //vels_out[i*M*M + j*M + k].w += 1.0*impulse/N;
  float3 pos = make_float3(verts[i], verts[i+1], verts[i+2]);
  if (pos.x > 0 && pos.x < M-1 &&
      pos.y > 0 && pos.y < M-1 &&
      pos.z > 0 && pos.z < M-1) {
    // for each of 8 nearest grid cells
    int i0,j0,k0,i1,j1,k1;
    float share;
    float d_buoy = 1024.0;
    float d_pres = 8192.0;
    // 0,0,0
    i0 = floor(pos.x); i1 = ceil(pos.x);
    j0 = floor(pos.y); j1 = ceil(pos.y);
    k0 = floor(pos.z); k1 = ceil(pos.z);
    // proportion given to each cell is equal to volume of the opposite quadrant
    share = abs((i1-pos.x) * (j1-pos.y) * (k1-pos.z));
    gvels[i0*M*M + j0*M + k0].z += d_buoy*dt*(times[i]-0.0)*share/N;
    gvels[i0*M*M + j0*M + k0].w += d_pres*dt*(times[i]-0.2)*share/N;
    // 0,0,1
    i0 = floor(pos.x); i1 = ceil(pos.x);
    j0 = floor(pos.y); j1 = ceil(pos.y);
    k1 = floor(pos.z); k0 = ceil(pos.z);
    // proportion given to each cell is equal to volume of the opposite quadrant
    share = abs((i1-pos.x) * (j1-pos.y) * (k1-pos.z));
    gvels[i0*M*M + j0*M + k0].z += d_buoy*dt*(times[i]-0.0)*share/N;
    gvels[i0*M*M + j0*M + k0].w += d_pres*dt*(times[i]-0.2)*share/N;
    // 0,1,0
    i0 = floor(pos.x); i1 = ceil(pos.x);
    j1 = floor(pos.y); j0 = ceil(pos.y);
    k0 = floor(pos.z); k1 = ceil(pos.z);
    // proportion given to each cell is equal to volume of the opposite quadrant
    share = abs((i1-pos.x) * (j1-pos.y) * (k1-pos.z));
    gvels[i0*M*M + j0*M + k0].z += d_buoy*dt*(times[i]-0.0)*share/N;
    gvels[i0*M*M + j0*M + k0].w += d_pres*dt*(times[i]-0.2)*share/N;
    // 0,1,1
    i0 = floor(pos.x); i1 = ceil(pos.x);
    j1 = floor(pos.y); j0 = ceil(pos.y);
    k1 = floor(pos.z); k0 = ceil(pos.z);
    // proportion given to each cell is equal to volume of the opposite quadrant
    share = abs((i1-pos.x) * (j1-pos.y) * (k1-pos.z));
    gvels[i0*M*M + j0*M + k0].z += d_buoy*dt*(times[i]-0.0)*share/N;
    gvels[i0*M*M + j0*M + k0].w += d_pres*dt*(times[i]-0.2)*share/N;
    // 1,0,0
    i1 = floor(pos.x); i0 = ceil(pos.x);
    j0 = floor(pos.y); j1 = ceil(pos.y);
    k0 = floor(pos.z); k1 = ceil(pos.z);
    // proportion given to each cell is equal to volume of the opposite quadrant
    share = abs((i1-pos.x) * (j1-pos.y) * (k1-pos.z));
    gvels[i0*M*M + j0*M + k0].z += d_buoy*dt*(times[i]-0.0)*share/N;
    gvels[i0*M*M + j0*M + k0].w += d_pres*dt*(times[i]-0.2)*share/N;
    // 1,0,1
    i1 = floor(pos.x); i0 = ceil(pos.x);
    j0 = floor(pos.y); j1 = ceil(pos.y);
    k1 = floor(pos.z); k0 = ceil(pos.z);
    // proportion given to each cell is equal to volume of the opposite quadrant
    share = abs((i1-pos.x) * (j1-pos.y) * (k1-pos.z));
    gvels[i0*M*M + j0*M + k0].z += d_buoy*dt*(times[i]-0.0)*share/N;
    gvels[i0*M*M + j0*M + k0].w += d_pres*dt*(times[i]-0.2)*share/N;
    // 1,1,0
    i1 = floor(pos.x); i0 = ceil(pos.x);
    j1 = floor(pos.y); j0 = ceil(pos.y);
    k0 = floor(pos.z); k1 = ceil(pos.z);
    // proportion given to each cell is equal to volume of the opposite quadrant
    share = abs((i1-pos.x) * (j1-pos.y) * (k1-pos.z));
    gvels[i0*M*M + j0*M + k0].z += d_buoy*dt*(times[i]-0.0)*share/N;
    gvels[i0*M*M + j0*M + k0].w += d_pres*dt*(times[i]-0.2)*share/N;
    // 1,1,1
    i1 = floor(pos.x); i0 = ceil(pos.x);
    j1 = floor(pos.y); j0 = ceil(pos.y);
    k1 = floor(pos.z); k0 = ceil(pos.z);
    // proportion given to each cell is equal to volume of the opposite quadrant
    share = abs((i1-pos.x) * (j1-pos.y) * (k1-pos.z));
    gvels[i0*M*M + j0*M + k0].z += d_buoy*dt*(times[i]-0.0)*share/N;
    gvels[i0*M*M + j0*M + k0].w += d_pres*dt*(times[i]-0.2)*share/N;
  }
}

void step_gpu(float* verts, float* times, float* colors,
              float4* gvel0, float4* gvel1, //float* gpres0, float* gpres1,
              const int N, const int M, int t,
              float flame_x, float flame_y) {
  int b = 8;
  dim3 gBlock(M/b,M/b,M/b);
  dim3 gThread(b,b,b);

  float visc = 0.10;
  // diffuse velocities
  diffuse<<<gBlock,gThread>>>(gvel1, gvel0, visc);
  //void** diffuse_args[3];
  //diffuse_args[0] = (void**)&gvel1; diffuse_args[1] = (void**)&gvel0; diffuse_args[2] = (void**)&visc;
  //hipLaunchCooperativeKernel((void*)diffuse, gBlock, gThread, (void**)diffuse_args);
  // Project
  //project<<<gBlock,gThread>>>(gvel1);
  // Pressure
  pressure<<<gBlock,gThread>>>(gvel0, gvel1);
  // Balance pressure
  //balance<<<gBlock,gThread>>>(gvel1);//, gvel1);
  // Advect Velocities
  advect<<<gBlock,gThread>>>(gvel1, gvel0, verts, times);
  // Ping the Pong
  //pingpong<<<Mblocks,512>>>(gvel1, gvel0);
  // Move Particles
  pstep<<<N/512,512>>>(gvel1, verts, times, colors, curandstate, flame_x, flame_y, decay_rate);
}

void step_cpu(float* verts, float* vels, float* times, float* colors, int N) {
#pragma omp parallel for
  for (int I=0; I < N; ++I) {
    int i = 3*I;
    verts[i  ] += vels[i  ];
    verts[i+1] += vels[i+1];
    verts[i+2] += vels[i+2] + 0.003*(1.0-times[I]);

    times[I] -= 0.0001;
    colors[i  ] = sqrt(times[I]);
    colors[i+1] = max(times[I]/1.125, 0.0);
    colors[i+2] = pow(times[I],2);
    if (times[I] <= 0.0) {
      times[I] = 1.0;
      verts[i  ] = M/2;
      verts[i+1] = M/2;
      verts[i+2] = M/2;
    }
  }
}
//////// SDL Init Function ////////

bool init(SDL_Window** window, SDL_GLContext* context)
{
  bool success = true;

  if (SDL_Init(SDL_INIT_VIDEO | SDL_INIT_TIMER) != 0)
  {
    cerr << "SDL failed to initialize: " << SDL_GetError() << endl;
    success = false;
  }

  *window = SDL_CreateWindow("Flame", 0,0, w,h, SDL_WINDOW_OPENGL | SDL_WINDOW_SHOWN | SDL_WINDOW_RESIZABLE);
  if (*window == NULL)
  {
    cerr << "SDL failed to create a window: " << SDL_GetError() << endl;
    success = false;
  }

  *context = SDL_GL_CreateContext(*window);
  if (*context == NULL)
  {
    cerr << "SDL failed to create OpenGL context: " << SDL_GetError() << endl;
    success = false;
  }

  //Vsync
  if (SDL_GL_SetSwapInterval(1) < 0)
  {
    cerr << "SDL could not set Vsync: " << SDL_GetError() << endl;
//    success = false;
  }

  cout << SDL_GetError() << endl;
  return success;
}

///////////////////////////////////

void display(SDL_Window* window, int r)
{
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  glEnable(GL_DEPTH_TEST);
  //glEnable(GL_CULL_FACE);

  //reshape(w,h);
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();

  //view angle
  ex = Sin(-th)*Cos(ph)*zoom;
  ey = Cos(-th)*Cos(ph)*zoom;
  ez = Sin(ph)*zoom;

  gluLookAt(ex+M/2,ey+M/2,ez+M/2, M/2,M/2,M/2, 0,0,Cos(ph));

  // lighting
  glEnable(GL_LIGHTING);
  float white[4]   = {1.0,1.0,1.0,1.0};
  float pos[4]     = {M/2+2.0, M/2-2.0, M/2+4.0, 1.0};
  float ambient[4] = {0.12, 0.15, 0.16, 1.0};
  float diffuse[4] = {0.65, 0.65, 0.60, 1.0};
  float specular[4]= {0.7, 0.7, 0.9, 1.0};
  float shininess  = 64;

  glColorMaterial(GL_FRONT_AND_BACK, GL_AMBIENT_AND_DIFFUSE);
  glEnable(GL_COLOR_MATERIAL);

  glEnable(GL_LIGHT0);
  glLightfv(GL_LIGHT0, GL_AMBIENT, ambient);
  glLightfv(GL_LIGHT0, GL_DIFFUSE, diffuse);
  glLightfv(GL_LIGHT0, GL_SPECULAR, specular);
  glLightfv(GL_LIGHT0, GL_POSITION, pos);

  glMaterialfv(GL_FRONT, GL_SHININESS, &shininess);
  glMaterialfv(GL_FRONT, GL_SPECULAR, white);

  // Object Rendering

  //glUseProgram(pixlight);
  //glColor3f(1.0,1.0,1.0);
  //ball(M/2,M/2,M/2, 0.25);

  if (changed) {
    if(hipSuccess != hipMemcpyAsync(verts, dverts, 3*N*sizeof(float), hipMemcpyDeviceToHost)) cout << "memcpy fail from " << dverts << " to " << verts << "\n";
    if(hipSuccess != hipMemcpyAsync(times, dtimes,   N*sizeof(float), hipMemcpyDeviceToHost)) cout << "memcpy fail from " << dtimes << " to " << times << "\n";
    if(hipSuccess != hipMemcpyAsync(colors,dcolors,3*N*sizeof(float), hipMemcpyDeviceToHost)) cout << "memcpy fail from " << dcolors << " to " << colors << "\n";
  }

  glUseProgram(shader);
  glDisable(GL_LIGHTING);
  glDisable(GL_DEPTH_TEST);
  glBindTexture(GL_TEXTURE_2D, starTexture);
  int id = glGetUniformLocation(shader, "star");
  if (id>=0) glUniform1i(id,0);
  // ^ current bound texture, star.bmp
  id = glGetUniformLocation(shader, "size");
  if (id>=0) glUniform1f(id,1.0);
  glEnable(GL_BLEND);
  glBlendFunc(GL_ONE,GL_ONE);

  glEnableClientState(GL_VERTEX_ARRAY);
  glEnableClientState(GL_COLOR_ARRAY);
  glVertexPointer(3,GL_FLOAT,0,verts);
  glColorPointer(3,GL_FLOAT,0,colors);

  //cout << "verts: " << verts[0] << "   \t" << verts[1] << "   \t" << verts[2] << endl;
  //cout << "color: " << colors[0]<< "   \t" << colors[1]<< "   \t" << colors[2] << endl;

  glDrawArrays(GL_POINTS,0,N);

  glDisable(GL_BLEND);
  glDisableClientState(GL_VERTEX_ARRAY);
  glDisableClientState(GL_COLOR_ARRAY);

  // Bounding Box
  glUseProgram(0);
  glEnable(GL_DEPTH_TEST);
  glBegin(GL_LINES);
  glColor3f(1.0,1.0,1.0);
  glVertex3f(0,0,0); glVertex3f(0,0,M);
  glVertex3f(0,0,M); glVertex3f(0,M,M);
  glVertex3f(0,M,M); glVertex3f(0,M,0);
  glVertex3f(0,M,0); glVertex3f(0,0,0);
  glVertex3f(0,0,0); glVertex3f(M,0,0);
  glVertex3f(0,0,M); glVertex3f(M,0,M);
  glVertex3f(0,M,M); glVertex3f(M,M,M);
  glVertex3f(0,M,0); glVertex3f(M,M,0);
  glVertex3f(M,0,0); glVertex3f(M,0,M);
  glVertex3f(M,0,M); glVertex3f(M,M,M);
  glVertex3f(M,M,M); glVertex3f(M,M,0);
  glVertex3f(M,M,0); glVertex3f(M,0,0);
  glEnd();
  glDisable(GL_DEPTH_TEST);


  //// DEBUG ////

  if (fieldlines) {
    // show velocities for debug purposes
    if (changed) {
      hipError_t err = hipMemcpyAsync(h_gvels, d_gvels[ping], 4*M*M*M*sizeof(float), hipMemcpyDeviceToHost); if (err != hipSuccess) {cout << "hipMemcpy failed: " << hipGetErrorString(err) << endl; quit = true;}
    }
    //cout << "Successfully copied Velocities from Device to Host\n";
    glUseProgram(0);
    glEnable(GL_DEPTH_TEST);
    glBegin(GL_LINES);
    for (int i=0; i < M; ++i) {
      for (int j=0; j < M; ++j) {
        for (int k=0; k < M; ++k) {
          glColor3f(1.0,0.5,0.0);
          glVertex3f(i, j, k);
          float x = h_gvels[4*(i*M*M + j*M + k)  ]*10.0;
          float y = h_gvels[4*(i*M*M + j*M + k)+1]*10.0;
          float z = h_gvels[4*(i*M*M + j*M + k)+2]*10.0;
          //float x = 0.0;
          //float y = 0.0;
          //float z = h_gvels[4*(i*M*M + j*M + k)+3]*10.0;
          glColor3f(0.5,0.0,0.0);
          glVertex3f(i+x, j+y, k+z);
        }
      }
    }
    glEnd();
    glDisable(GL_DEPTH_TEST);
  }

  //// show other values for debug purposes
  //hipError_t err = hipMemcpy(h_gtemp, d_gtemp[ping], M*M*M*sizeof(float), hipMemcpyDeviceToHost); if (err != hipSuccess) {cout << "hipMemcpy failed: " << hipGetErrorString(err) << endl; quit = true;}
  //glUseProgram(0);
  //glBegin(GL_LINES);
  //for (int i=0; i < M; ++i) {
  //  for (int j=0; j < M; ++j) {
  //    for (int k=0; k < M; ++k) {
  //      glColor3f(1.0,1.0,1.0);
  //      glVertex3f(i, j, k);
  //      float z = h_gtemp[i*M*M + j*M + k]*10.0;
  //      glColor3f(0.1,0.1,0.1);
  //      glVertex3f(i, j, k+z);
  //    }
  //  }
  //}
  //glEnd();

  changed = false;

  //swap the buffers
  glFlush();
  SDL_GL_SwapWindow(window);
}

void physics(int r)
{
  const Uint8* state = SDL_GetKeyboardState(NULL);
  keyboard(state);

  //adjust the eye position
  th += dth;
  ph += dph;
  zoom = zoom<2.0?2.0:zoom+dzoom;

  // Step Flame Animation ////
  if (!stepmode && !Pause) {
    if (gpu) {
      if(hipSuccess != hipMemcpyAsync(verts, dverts, 3*N*sizeof(float), hipMemcpyDeviceToHost)) cout << "memcpy fail from " << dverts << " to " << verts << "\n";
      if(hipSuccess != hipMemcpyAsync(times, dtimes,   N*sizeof(float), hipMemcpyDeviceToHost)) cout << "memcpy fail from " << dtimes << " to " << times << "\n";
      if(hipSuccess != hipMemcpyAsync(colors,dcolors,3*N*sizeof(float), hipMemcpyDeviceToHost)) cout << "memcpy fail from " << dcolors << " to " << colors << "\n";
      ////cout << "successfully copied Particles from Device to Host" << endl;
      //for (int I=0; I < N; ++I) {
      //  int i = I*3;
      //  if (times[I]   < 0.0 ||
      //  verts[i]   < 0.0 ||
      //  verts[i]   > M   ||
      //  verts[i+1] < 0.0 ||
      //  verts[i+1] > M   ||
      //  verts[i+2] < 0.0 ||
      //  verts[i+2] > M  ) {
      //    times[I]   = 1.0f;
      //    verts[i  ] = 8*((float)rand()/(float)RAND_MAX-0.5) + M/2;
      //    verts[i+1] = 8*((float)rand()/(float)RAND_MAX-0.5) + M/2;
      //    verts[i+2] = 8*((float)rand()/(float)RAND_MAX-0.5) + M/2;
      //  }
      //}
      //if (true) { //{*/r < 10000) {
      //  if(hipSuccess != hipMemcpy(h_gvels, d_gvels[ping], 4*M*M*M*sizeof(float), hipMemcpyDeviceToHost)) cout << "memcpy fail from " << dcolors << " to " << colors << "\n";
      //  h_gvels[4*((M/2)*M*M + (M/2)*M + (M/2))+2] = 1.0;
      //  //h_gvels[4*((M/2)*M*M + (M/2)*M + (M/2))+2] += 1.0;
      //  //h_gvels[4*((M/2-1)*M*M + (M/2)*M + (M/2))+0] = -1.0;
      //  //h_gvels[4*((M/2+1)*M*M + (M/2)*M + (M/2))+0] = 1.0;
      //  //h_gvels[4*((M/2)*M*M + (M/2-1)*M + (M/2))+1] = -1.0;
      //  //h_gvels[4*((M/2)*M*M + (M/2+1)*M + (M/2))+1] = 1.0;
      //  //h_gvels[4*((M/2)*M*M + (M/2)*M + (M/2-1))+2] = -1.0;
      //  //h_gvels[4*((M/2)*M*M + (M/2)*M + (M/2+1))+2] = 1.0;
      //  if(hipSuccess != hipMemcpy(d_gvels[ping], h_gvels, 4*M*M*M*sizeof(float), hipMemcpyHostToDevice)) cout << "memcpy fail from " << verts << " to " << dverts << "\n";
      //}
      ////if(hipSuccess != hipMemcpy(d_gpres[0], zeros,   M*M*M*sizeof(float), hipMemcpyHostToDevice)) cout << "failure to memcpy: " << endl;
      ////if(hipSuccess != hipMemcpy(d_gpres[1], zeros,   M*M*M*sizeof(float), hipMemcpyHostToDevice)) cout << "failure to memcpy: " << endl;
      //if(hipSuccess != hipMemcpy(dverts, verts, 3*N*sizeof(float), hipMemcpyHostToDevice)) cout << "memcpy fail from " << verts << " to " << dverts << "\n";
      //if(hipSuccess != hipMemcpy(dtimes, times,   N*sizeof(float), hipMemcpyHostToDevice)) cout << "memcpy fail from " << times << " to " << dtimes << "\n";
      //if(hipSuccess != hipMemcpy(dcolors,colors,3*N*sizeof(float), hipMemcpyHostToDevice)) cout << "memcpy fail from " << colors << " to " << dcolors << "\n";
      ////cout << "successfully copied Particles from Host to Device" << endl;
      step_gpu(dverts, dtimes, dcolors,
               d_gvels[ping], d_gvels[pong],// d_gtemp[ping], d_gtemp[pong], d_gdens[ping], d_gdens[pong], d_gpres[0], d_gpres[1], d_diverge,
               //s_gvels[pong], s_gtemp[pong], s_gdens[pong],
               N, M, r, flame_x, flame_y);
      ping = pong;
      pong = 1-pong;
    }
    else {
      step_cpu(verts, pvels, times, colors, N);
    }
    changed = true;
  }
  ////////////////////////////
}


void reshape(int width, int height)
{
  w = width;
  h = height;
  //new aspect ratio
  double w2h = (height > 0) ? (double)width/height : 1;
  //set viewport to the new window
  glViewport(0,0 , width,height);

  //switch to projection matrix
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();

  //adjust projection
  //glOrtho(-w2h, w2h, -1, 1, -1, 1);
  gluPerspective(60, w2h, 1.0, 4*M);

  //switch back to model matrix
  glMatrixMode(GL_MODELVIEW);
  glLoadIdentity();
}

static void Reverse(void* x,const int n)
{
   int k;
   char* ch = (char*)x;
   for (k=0;k<n/2;k++)
   {
      char tmp = ch[k];
      ch[k] = ch[n-1-k];
      ch[n-1-k] = tmp;
   }
}
int LoadTexture(const char* file) {
  unsigned int   texture;    // Texture name
  FILE*          f;          // File pointer
  unsigned short magic;      // Image magic
  int            dx,dy;
  unsigned int   size;       // Image dimensions
  unsigned short nbp,bpp;    // Planes and bits per pixel
  unsigned char* image;      // Image data
  unsigned int   k;          // Counter
  int            max;        // Maximum texture dimensions

  //  Open file
  f = fopen(file,"rb");
  if (!f) fprintf(stderr,"Cannot open file %s\n",file);
  //  Check image magic
  if (fread(&magic,2,1,f)!=1) fprintf(stderr,"Cannot read magic from %s\n",file);
  if (magic!=0x4D42 && magic!=0x424D) fprintf(stderr,"Image magic not BMP in %s\n",file);
  //  Seek to and read header
  if (fseek(f,16,SEEK_CUR) || fread(&dx ,4,1,f)!=1 || fread(&dy ,4,1,f)!=1 ||
      fread(&nbp,2,1,f)!=1 || fread(&bpp,2,1,f)!=1 || fread(&k,4,1,f)!=1)
    fprintf(stderr,"Cannot read header from %s\n",file);
  //  Reverse bytes on big endian hardware (detected by backwards magic)
  if (magic==0x424D)
  {
     Reverse(&dx,4);
     Reverse(&dy,4);
     Reverse(&nbp,2);
     Reverse(&bpp,2);
     Reverse(&k,4);
  }

  dx = abs(dx);
  dy = abs(dy);

  //  Check image parameters
  glGetIntegerv(GL_MAX_TEXTURE_SIZE,&max);
  if (dx<1 || dx>max) fprintf(stderr,"%s image width %d out of range 1-%d\n",file,dx,max);
  if (dy<1 || dy>max) fprintf(stderr,"%s image height %d out of range 1-%d\n",file,dy,max);
  if (nbp!=1)  fprintf(stderr,"%s bit planes is not 1: %d\n",file,nbp);
  if (bpp!=24) fprintf(stderr,"%s bits per pixel is not 24: %d\n",file,bpp);
  if (k!=0)    fprintf(stderr,"%s comdenssed files not supported\n",file);
#ifndef GL_VERSION_2_0
  //  OpenGL 2.0 lifts the restriction that texture size must be a power of two
  for (k=1;k<dx;k*=2);
  if (k!=dx) fprintf(stderr,"%s image width not a power of two: %d\n",file,dx);
  for (k=1;k<dy;k*=2);
  if (k!=dy) fprintf(stderr,"%s image height not a power of two: %d\n",file,dy);
#endif

  //  Allocate image memory
  size = 3*dx*dy;
  image = (unsigned char*) malloc(size);
  if (!image) fprintf(stderr,"Cannot allocate %d bytes of memory for image %s\n",size,file);
  //  Seek to and read image
  if (fseek(f,20,SEEK_CUR) || fread(image,size,1,f)!=1) fprintf(stderr,"Error reading data from image %s\n",file);
  fclose(f);
  //  Reverse pvels   (BGR -> RGB)
  for (k=0;k<size;k+=3)
  {
     unsigned char temp = image[k];
     image[k]   = image[k+2];
     image[k+2] = temp;
  }

  //  Sanity check
  //ErrCheck("LoadTexBMP");
  //  Generate 2D texture
  glGenTextures(1,&texture);
  glBindTexture(GL_TEXTURE_2D,texture);
  //  Copy image
  glTexImage2D(GL_TEXTURE_2D,0,3,dx,dy,0,GL_RGB,GL_UNSIGNED_BYTE,image);
  if (glGetError()) fprintf(stderr,"Error in glTexImage2D %s %dx%d\n",file,dx,dy);
  //  Scale linearly when image size doesn't match
  glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_MAG_FILTER,GL_LINEAR);
  glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_MIN_FILTER,GL_LINEAR);

  //  Free image memory
  free(image);
  //  Return texture name
  return texture;

}

// Per frame keyboard input here, per keydenss input in main()
void keyboard(const Uint8* state)
{
  //if (state[SDL_SCANCODE_ESCAPE])
  //  quit = true;

  if (state[SDL_SCANCODE_LEFT])
    dth = -0.75;
  else if (state[SDL_SCANCODE_RIGHT])
    dth = 0.75;
  else
    dth = 0;

  if (state[SDL_SCANCODE_DOWN])
    dph = -0.75;
  else if (state[SDL_SCANCODE_UP])
    dph = 0.75;
  else
    dph = 0;

  if (state[SDL_SCANCODE_Z])
    dzoom = -0.10;
  else if (state[SDL_SCANCODE_X])
    dzoom = 0.10;
  else
    dzoom = 0;
}

// all user interaction goes here
bool handleEvents()
{
  SDL_Event event;

  while (SDL_PollEvent(&event))
  {
    switch(event.type)
    {
      case SDL_QUIT:
        return true;

      case SDL_MOUSEMOTION:
        if (flame_moving)
        {
          float dx_f = ( Cos(th)*event.motion.xrel + Sin(th)*event.motion.yrel) / M * 2.0;
          float dy_f = (-Sin(th)*event.motion.xrel + Cos(th)*event.motion.yrel) / M * 2.0;
          flame_x = min(max(flame_x - dx_f, 1.0), M-2.0);
          flame_y = min(max(flame_y + dy_f, 1.0), M-2.0);
          //cout << flame_x << "\t" << flame_y << endl;
        }
        if (rotating)
        {
          th -= event.motion.xrel/(2.0*M_PI);
          ph += event.motion.yrel/(2.0*M_PI);
        }
        break;

      case SDL_MOUSEWHEEL:
        if (event.wheel.y > 0)
          zoom /= pow(2.0, 1.0/8.0);
        else if (event.wheel.y < 0)
          zoom *= pow(2.0, 1.0/8.0);
        break;

      case SDL_MOUSEBUTTONDOWN:
        switch (event.button.button)
        {
          case SDL_BUTTON_LEFT:
            flame_moving = true;
            break;

          case SDL_BUTTON_RIGHT:
            rotating = true;
            break;
        }
        break;

      case SDL_MOUSEBUTTONUP:
        switch (event.button.button)
        {
          case SDL_BUTTON_LEFT:
            flame_moving = false;
            break;

          case SDL_BUTTON_RIGHT:
            rotating = false;
            break;
        }
        break;

      case SDL_KEYDOWN:
        switch (event.key.keysym.scancode)
        {
          case SDL_SCANCODE_Q:
            return true;

          case SDL_SCANCODE_SPACE:
            Pause = 1 - Pause;
            break;

          case SDL_SCANCODE_M:
            stepmode = !stepmode;
            break;

          case SDL_SCANCODE_G:
            gpu = !gpu;
            break;

          case SDL_SCANCODE_V:
            fieldlines = !fieldlines;
            break;

          case SDL_SCANCODE_COMMA:
            tick_period *= pow(2,0.25);
            break;

          case SDL_SCANCODE_PERIOD:
            tick_period /= pow(2,0.25);
            break;

          case SDL_SCANCODE_LEFTBRACKET:
            decay_rate *= pow(2,0.25);
            break;

          case SDL_SCANCODE_RIGHTBRACKET:
            decay_rate /= pow(2,0.25);
            break;

          default:
            break;
        }

      case SDL_WINDOWEVENT:
        if (event.window.event == SDL_WINDOWEVENT_SIZE_CHANGED)
        {
          //cerr << event.window.data1 << " " << event.window.data2 << endl;
          reshape(event.window.data1, event.window.data2);
        }
        break;
    }
  }
  return false;
}

int main(int argc, char *argv[])
{
  //SDL Window/OpenGL Context
  SDL_Window* window = NULL;
  SDL_GLContext context;

  //Initialize
  if (init(&window, &context) != true)
  {
    cerr << "Shutting Down\n";
    return 1;
  }

  h_gvels = new float[4*M*M*M];
  //h_gtemp = new float[M*M*M];

  verts = new float[3*N];
  pvels = new float[3*N];
  times = new float[N];
  colors= new float[3*N];
  //memset(verts, 0.0, 3*N*sizeof(float));
  //memset(pvels  ,0.0, 3*N*sizeof(float));
  for (int i=0; i < 3*N; i += 3) {
    verts[i  ] = 8*((float)rand()/(float)RAND_MAX - 0.5) + M/2;
    verts[i+1] = 8*((float)rand()/(float)RAND_MAX - 0.5) + M/2;
    verts[i+2] = 8*((float)rand()/(float)RAND_MAX - 0.5) + M/2;
    pvels[i  ] = ((float)rand()/(float)RAND_MAX - 0.5)/1000.0;
    pvels[i+1] = ((float)rand()/(float)RAND_MAX - 0.5)/1000.0;
    pvels[i+2] = ((float)rand()/(float)RAND_MAX - 0.5)/1000.0;
    //verts[i] = 0;
    //pvels[i] = 0;
  }
  for (int i=0; i < N; ++i)
    //times[i]= ((float)rand()/(float)RAND_MAX);
    times[i]= (float)(i)/N;

  //allocate particle  and grid arrays
  if(hipSuccess != hipMalloc(&dverts, 3*N*sizeof(float))) cout << "failure to allocate\n";
  if(hipSuccess != hipMalloc(&dpvels, 3*N*sizeof(float))) cout << "failure to allocate\n";
  if(hipSuccess != hipMalloc(&dtimes,   N*sizeof(float))) cout << "failure to allocate\n";
  if(hipSuccess != hipMalloc(&dcolors,3*N*sizeof(float))) cout << "failure to allocate\n";

  if(hipSuccess != hipMalloc(&d_gvels[0],4*M*M*M*sizeof(float))) cout << "failure to allocate\n";
  if(hipSuccess != hipMalloc(&d_gvels[1],4*M*M*M*sizeof(float))) cout << "failure to allocate\n";
  //if(hipSuccess != hipMalloc(&d_gtemp[0],  M*M*M*sizeof(float))) cout << "failure to allocate\n";
  //if(hipSuccess != hipMalloc(&d_gtemp[1],  M*M*M*sizeof(float))) cout << "failure to allocate\n";
  //if(hipSuccess != hipMalloc(&d_gdens[0],  M*M*M*sizeof(float))) cout << "failure to allocate\n";
  //if(hipSuccess != hipMalloc(&d_gdens[1],  M*M*M*sizeof(float))) cout << "failure to allocate\n";
  //if(hipSuccess != hipMalloc(&d_gpres[0],  M*M*M*sizeof(float))) cout << "failure to allocate\n";
  //if(hipSuccess != hipMalloc(&d_gpres[1],  M*M*M*sizeof(float))) cout << "failure to allocate\n";
  //if(hipSuccess != hipMalloc(&d_diverge,   M*M*M*sizeof(float))) cout << "failure to allocate\n";

  //memset(zeros, 0.0, 4*M*M*M*sizeof(float));
  hipError_t err;
  err = hipMemcpy(d_gvels[0], zeros, 4*M*M*M*sizeof(float), hipMemcpyHostToDevice); if (err) cout << "failure to memcpy: " << hipGetErrorString(err) << endl;
  err = hipMemcpy(d_gvels[1], zeros, 4*M*M*M*sizeof(float), hipMemcpyHostToDevice); if (err) cout << "failure to memcpy: " << hipGetErrorString(err) << endl;
  //err = hipMemcpy(d_gtemp[0], zeros,   M*M*M*sizeof(float), hipMemcpyHostToDevice); if (err) cout << "failure to memcpy: " << hipGetErrorString(err) << endl;
  //err = hipMemcpy(d_gtemp[1], zeros,   M*M*M*sizeof(float), hipMemcpyHostToDevice); if (err) cout << "failure to memcpy: " << hipGetErrorString(err) << endl;
  //err = hipMemcpy(d_gpres[0], zeros,   M*M*M*sizeof(float), hipMemcpyHostToDevice); if (err) cout << "failure to memcpy: " << hipGetErrorString(err) << endl;
  //err = hipMemcpy(d_gpres[1], zeros,   M*M*M*sizeof(float), hipMemcpyHostToDevice); if (err) cout << "failure to memcpy: " << hipGetErrorString(err) << endl;
  //err = hipMemcpy(d_diverge,  zeros,   M*M*M*sizeof(float), hipMemcpyHostToDevice); if (err) cout << "failure to memcpy: " << hipGetErrorString(err) << endl;
  //
  //err = hipMemcpy(d_gdens[0], zeros,   M*M*M*sizeof(float), hipMemcpyHostToDevice); if (err) cout << "failure to memcpy: " << hipGetErrorString(err) << endl;
  //err = hipMemcpy(d_gdens[1], zeros,   M*M*M*sizeof(float), hipMemcpyHostToDevice); if (err) cout << "failure to memcpy: " << hipGetErrorString(err) << endl;
  if(hipSuccess != hipMemcpy(dverts, verts, 3*N*sizeof(float), hipMemcpyHostToDevice)) cout << "memcpy fail\n";
  if(hipSuccess != hipMemcpy(dtimes, times,   N*sizeof(float), hipMemcpyHostToDevice)) cout << "memcpy fail\n";
  if(hipSuccess != hipMemcpy(dcolors,colors,3*N*sizeof(float), hipMemcpyHostToDevice)) cout << "memcpy fail\n";
  if(hipSuccess != hipMemcpy(dpvels, pvels, 3*N*sizeof(float), hipMemcpyHostToDevice)) cout << "memcpy fail\n";

  //allocate Cuda random state
  if(hipSuccess != hipMalloc(&curandstate, 3*N*sizeof(hiprandState_t))) cout << "failure to allocate state\n";
  init_rand_state<<<3*N/512,512>>>(curandstate);

  if (err) quit = true;

  //////////////////////////////////////////////////////

  starTexture = LoadTexture("star.bmp");
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
  glTexEnvi(GL_TEXTURE_2D, GL_TEXTURE_ENV_MODE, GL_REPLACE);

  //Timing
  float r = 0;
  float dr = 0;
  float oldr = 0;
  //int Pause = 0;
  int frames = 0;

  //shader
  shader = CreateShaderProgGeom((char*)"flame.vert", (char*)"flame.geom", (char*)"flame.frag");
  pixlight = CreateShaderProg((char*)"pixlight.vert", (char*)"pixlight.frag");

  reshape(w,h);

  int startuptime = SDL_GetTicks();
  oldr = startuptime;

  ////////Main Loop////////
  //bool quit = false;
  try {
    while (!quit)
    {
      //cout << "handling events\n";
      quit = handleEvents();

      ////Physics Timing////
      r = SDL_GetTicks();
      dr += r - oldr;
      while (dr >= tick_period)
      {
        // 1000/8 = 125 updates per second
        physics(r);
        dr -= tick_period;
      }
      oldr = r;
      display(window, r);
      frames += 1;
      //quit = true;
    }
  }
  catch (...) {cout << "catch block\n";}

  cout << "Shutting Down\n";
  cout << "average framerate: " << 1000*(float)frames/(r - startuptime) << endl;

  hipFree(dverts);
  hipFree(dpvels);
  hipFree(dtimes);
  hipFree(dcolors);
  hipFree(d_gvels[0]);
  hipFree(d_gvels[1]);
  hipFree(curandstate);
  //hipFree(d_gtemp[0]);
  //hipFree(d_gtemp[1]);
  //hipFree(d_gdens[0]);
  //hipFree(d_gdens[1]);
  //hipFree(d_gpres[0]);
  //hipFree(d_gpres[1]);
  //hipFree(d_diverge);
  delete verts;
  delete pvels;
  delete times;
  delete colors;

  delete h_gvels;
  //delete h_gtemp;

  SDL_Quit();

  return 0;
}
